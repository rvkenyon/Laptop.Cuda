#include "hip/hip_runtime.h"
//#define WIN32 1

#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
#include "stdio.h"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>


#define THREADS 512
//#define Gy 32 //grid y dimension
//#define Gx 1920 //grid x dimension
//#define Xt 1 //thread x dimension
//#define Yt 64 //thread y dimension
//#define Fx 2000 //number of frames
typedef struct{
	int x;
	int y;
	} twoD;


typedef struct{
	unsigned int win;
	float sDev;
	} pixelLoc;

typedef struct{
	unsigned int loc_Wind1;
	unsigned int loc_Wind2;
	float loc_corrCoef;
	} PixelxCor;

int const Gy = 32; //grid y dimension
int const Gx = 4; //grid x dimension
int const Xt = 512; //thread x dimension
//int const Gx = 1920; //grid x dimension
//int const Xt = 1; //thread x dimension
int const Yt = 1; //32//thread y dimension
int const Fx = 2000; //number of frames
int const h_Wsize = 50;
int const devThres = 15;

using namespace std;

//this where each thread takes a different window to Xcorr
//__global__ void XcrossCUDA(int* d_Pixels, pixelLoc* d_PL, float* d_Cor, int N, int corCount, int Wsize)
//{
//	int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
//	//	int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
//
//	float x1, x2, SumPt2, Sum_X1X2, window[120],sdev1,sdev2;
//	int winStart, window1,window2, Index,temp2,temp3; //change yIdx and xIdx
//	//	float xcorrValues[16000];
//
//	// find local point only for xcorr with window
//	if(xIdx < N-1 && d_PL[xIdx].win > 0)//|| window1 > 0) //not at end of file
//	{
//		winStart = xIdx;
//		temp3 = xIdx; //index from "points to correlate" array d_PL
//		window1 = d_PL[winStart].win;
//		//load window for xcorr
//		sdev1 = d_PL[winStart].sDev;
//
//		//temp2 = ((temp3) * (temp3-1))/2;
//		//Index = (N * temp3) - temp2; //this needs to be checked
//		//calculate the offset to write the output data file [N * I - (N(N-1)/2)]
//		//if(xIdx == 0)
//		//	Index = 0;
//		//else 
//		//	{
//		//	temp2 = ;
//		Index = corCount - ((N-xIdx) * (N-xIdx - 1))/2; //this needs to be checked
//		//	}
//
//		//get pixel values for correlation's Master window
//		for(int n = 0; n < Wsize; n++) 
//			window[n] = d_Pixels[window1 + n];
//
//		//now get windows for other points in the correlation
//		for(int i = 0; i < N - xIdx; i++)
//		{
//			window2 = d_PL[winStart+i].win;
//			sdev2 = d_PL[winStart+i].sDev;
//			//			if(sdev2 == 0 || sdev1 == 0) 
//			//			continue;
//			//if(window2 < 0)
//			//	break;
//			//find data start point for windows in silo
//
//			//if point is valid then begin correlations
//			x1 = x2 = Sum_X1X2 = 0.;
//
//			// do the actual cross correlation now
//			for (int l = 0; l < Wsize; l++)
//			{
//				SumPt2 = d_Pixels[window2 + l];
//				x1 += window[l];
//				x2 += SumPt2;
//				Sum_X1X2 += window[l] * SumPt2;
//			}
//			d_Cor[i + Index] = ((Sum_X1X2 - x1 * x2/Wsize)/(Wsize - 1)/sdev2/sdev1);	
//		} //end of correlation calculation
//	} // end of finding windows inside pixel silo
//}

//this where thread acts on the same window to Xcorr
__global__ void XcrossCUDA_same(int* d_Pixels, pixelLoc* d_PL, PixelxCor* d_Cor, int X, int corCount, int Wsize)
	{
	__shared__  int window[h_Wsize];
	//__shared__ 
	//here d_Cor is on Host not Device
	unsigned int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
	float sdev1;
	float x1, x2, SumPt2, Sum_X1X2, sdev2;
	unsigned int window2, Index,window1,winStart,j; //change yIdx and xIdx

	// find local point only for xcorr with window
	if(xIdx < X-1)
		{
		for(winStart = 0, j = xIdx; xIdx < X-1 - winStart; j = xIdx, winStart++) //increment through all PL data points
			{
			//		if(xIdx < N-i) //not at end of file
			//	{
			Index = corCount - ((X-winStart) * (X-winStart - 1))/2; //this needs to be checked
			//			winStart = i; //index of the window
			window1 = d_PL[winStart].win;
			sdev1 = d_PL[winStart].sDev;
			//get pixel values for correlation's Master window
			if(threadIdx.x == 0)
				{
				for(int ib = 0; ib < h_Wsize; ib++)
					window[ib] = d_Pixels[window1 + ib];
				}
			//if(threadIdx.x < Wsize)
			//	window[threadIdx.x] = d_Pixels[window1 + threadIdx.x];
			__syncthreads();

			//roll through all the data for this window
			while(j < X-1-winStart)
				{
				window2 = d_PL[winStart+j].win;
				sdev2 = d_PL[winStart+j].sDev;

				//if point is valid then begin correlations
				x1 = x2 = Sum_X1X2 = 0.;

				// do the actual cross correlation now
				for (int l = 0; l < Wsize; l++)
					{
					SumPt2 = d_Pixels[window2 + l];
					x1 += window[l];
					x2 += SumPt2;
					Sum_X1X2 += window[l] * SumPt2;
					}
				d_Cor[j + Index].loc_corrCoef = ((Sum_X1X2 - x1 * x2/Wsize)/(Wsize - 1)/sdev1/sdev2);	
				d_Cor[j + Index].loc_Wind1 = window1;	
				d_Cor[j + Index].loc_Wind2 = window2;	

				j += gridDim.x * blockDim.x;
				}
			__syncthreads(); //need this so wndow not changed while still in use.
			}
		}
	}


__global__ void StdDev(int* d_Pixels, pixelLoc* d_PL,  int Wsize, int frames,  int yTotal, twoD numProcThds, int devThres)
	{
	unsigned int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

	float temp, x1=0.f, x2=0.f;
	unsigned int xyStart; //where to start reading the window
	unsigned int outStart;   //output file indexing
	if(xIdx < numProcThds.x && yIdx < numProcThds.y)
		{
		while(yIdx < yTotal)
			{
			//Calculate Standard Deviation per window within XY pixel silo
			xyStart = xIdx + frames * yIdx;
			outStart = xIdx + numProcThds.x * yIdx;
			//			outStart = xIdx + gridDim.x * blockDim.x * yIdx;
			x1 = x2 = 0.;
			for(int i = 0; i < Wsize; i++)
				{
				temp = d_Pixels[xyStart + i];
				x1 += temp;
				x2 += temp * temp; 
				}
			temp = sqrtf((x2 - x1*x1/Wsize)/(Wsize-1));
			d_PL[outStart].win = xyStart;
			if(temp > devThres)
				d_PL[outStart].sDev = temp;
			else
				d_PL[outStart].sDev = 0.0f;
			yIdx += gridDim.y*blockDim.y;
			}
		}
	}


int main()
	{
	int const count = Fx*Gy*Yt; //Fx=MaxX, Gy * Yt = maxY for data file
	twoD numProcThds;
	numProcThds.x = Fx - h_Wsize; //used in Stdev kernel for total number threads X direction
	numProcThds.y = Gy*Yt; //used in Stdev kernel for total number threads Y direction
	int const imageX = 17; //size of Image used ... columns
	int const imageY = 13*5; //size of Image used ... rows
	int const totalPixs = imageX * imageY; //total pixel number for image
	int  readSize = Fx * totalPixs; //total memory size of all data
	int i = 0,N;
	int size_file=0;
	int Xloc1, Yloc1, Floc1; //used for X,Y,Frame for Point 1
	int Xloc2, Yloc2, Floc2; //used for X,Y,Frame for Point 2


	int procsrTot = numProcThds.x*numProcThds.y;
	pixelLoc *h_PL = new pixelLoc[readSize]; //used to hold Stdev values
	int *h_Pixels = new int[readSize]; //used to hold pixel values
	int  asd=sizeof(pixelLoc);
	pixelLoc *d_PL; //device version of h_PL
	int *d_Pixels;  //device version of h_Pixels
	PixelxCor *d_Cor;  //device version of h_Cor
	int deviceCount;
	int frames = Fx;
	int yTotal;
	int dev = 0;

	//this MUST be here; flags must be set before any
	//Cuda calls made; if Host Memory use by Device is used!!
	hipSetDeviceFlags(hipDeviceMapHost);


	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) 
		{
		fprintf(stderr, "error: no devices supporting CUDA.\n");
		exit(EXIT_FAILURE);
		}

	hipSetDevice(dev);

	hipDeviceProp_t devProps;
	if (hipGetDeviceProperties(&devProps, dev) == 0)
		{
		printf("Using device %d:\n", dev);
		printf("%s; global mem: %uB; compute v%d.%d; clock: %d kHz\n",
			devProps.name, (long)devProps.totalGlobalMem, 
			(int)devProps.major, (int)devProps.minor, 
			(int)devProps.clockRate);
		}
	int const gridLimit = devProps.maxGridSize[0];
	int  thredMax = 128; //devProps.maxThreadsPerBlock;

	const dim3 blockSize(thredMax, Yt, 1);  //TODO
	const dim3 gridSize(((Fx-1)/thredMax)+1,Gy, 1);  //TODO
	//int Tot_NumThreads;
	//int BlockWidth;
	//first = Tot_NumThreads/BlockWidth;
	//second = BlockWidth;//threads per block

	//read a binary file of data
	std::ifstream fin("c:/data/file_.bin", std::ios::binary);
	fin.read(reinterpret_cast<char*>(h_Pixels), sizeof(int) * readSize);
	fin.close();

	//FILE* file;
	//file = fopen("c:/data/file_name20.txt", "r");
	//if(file == 0)
	//	{
	//	printf("bad file name\n");
	//	exit(0);
	//	}
	//		while (!feof (file))
	////	for(int i = 0; i < readSize; i++)
	//		{  
	//		fscanf(file, "%E", &temp);
	//		h_Pixels[ib++] = int(temp);
	//		size_file++;
	//		}

	//	readSize = size_file;
	//   std::ofstream fout("c:/data/file_.bin", std::ios::binary);
	//      fout.write(reinterpret_cast<char*>(h_Pixels), sizeof(int) * readSize);
	//fout.close();
	//readSize = size_file;
	//std::ifstream fin("c:/data/file_.bin", std::ios::binary);
	//fin.read(reinterpret_cast<char*>(h_Pixels), sizeof(int) * readSize);
	//fin.close();

	//for(int i=0; i < readSize; i++)
	//	if(m[i] != h_Pixels[i])
	//		cout<<"wrong"<<endl;

	cout<<"Prior to addition: "<<endl;

	for(int i = 0; i < 10; i++)
		{
		cout<<h_Pixels[i]<<endl;
		}

	//	size_file = Fx * 5000;
	//	readSize = Fx * 5000;

	//hipMalloc((void**) &d_PL, sizeof(pixelLoc) * readSize);
	//	hipMemset((void*) d_PL, 0, sizeof(pixelLoc) * readSize);
	//	checkCudaErrors(hipMalloc((void**) &d_Pixels, sizeof(int) * readSize));

	//allocate memory space and copy data to device
	hipMalloc((void**) &d_PL, sizeof(pixelLoc) * readSize);
	hipMemset((void*) d_PL, 0, sizeof(pixelLoc) * readSize);
	hipMalloc((void**) &d_Pixels, sizeof(int) * readSize);
	hipMemcpy((void*) d_Pixels, h_Pixels, sizeof(int) * readSize, hipMemcpyHostToDevice);

	//run kernel for finding Standard Deviation of data
	StdDev<<<gridSize, blockSize>>>(d_Pixels, d_PL, h_Wsize, frames, totalPixs, numProcThds, devThres);

	//wait for all to finish and copy data to host
	hipDeviceSynchronize(); 
	hipGetLastError();
	hipMemcpy(h_PL, d_PL, sizeof(pixelLoc) * readSize, hipMemcpyDeviceToHost);

	//compress list of points, removing points below threshold 
	int j = 0;
	for(int i = 0; i < readSize; i++)
		{
		//if(h_PL[i].sDev < 1 && h_PL[i].win != -1)
		//	{
		//	cout<<"std = "<<h_PL[i].sDev<<"   "<<h_PL[i].win<<endl;
		//	}
		if(h_PL[i].sDev > 0)
			{
			h_PL[j++] = h_PL[i];
			}
		}
	N = j;
	//	hipFree(d_Pixels);
	//	hipMalloc((void**) &d_Pixels, sizeof(int) * readSize);
	hipFree(d_PL);
	hipMalloc((void**) &d_PL, sizeof(pixelLoc) * N);

	int const N1 = N +1;
	unsigned int const corSize = N1*(N1-1)/2;
	PixelxCor *h_Cor;// = new PixelxCor[corSize];
	//	hipMalloc((void**) &d_Cor, sizeof(PixelxCor) * corSize);

	//use memory on Host for Kernel not Device due to Size of Array
	hipHostAlloc((void**)&h_Cor, sizeof(PixelxCor) * corSize, hipHostMallocMapped);

	//get the address for Kernel write to output array
	hipHostGetDevicePointer(&d_Cor, h_Cor, 0);

	//do the regular stuff for passing arrays to Kernel
	//	hipMemcpy((void*) d_Pixels, h_Pixels, sizeof(int) * readSize, hipMemcpyHostToDevice);
	hipMemcpy((void*) d_PL, h_PL, sizeof(pixelLoc) * N, hipMemcpyHostToDevice);

	//int *Indexing = new int[300000];
	//for(int idx = 0; idx < N; idx++)
	//	Indexing[idx] = corSize - ((N1-idx) * (N1-idx - 1))/2; //this needs to be checked

	//now do xcorrelation
	int  blocks = (N+thredMax-1)/thredMax;
	if(blocks > gridLimit) blocks = gridLimit;

	XcrossCUDA_same<<<blocks, thredMax>>>(d_Pixels, d_PL,  d_Cor, N1, corSize, h_Wsize);

	hipDeviceSynchronize(); 
	hipGetLastError();
	delete[] h_Pixels;
	hipFree(d_Pixels);

	//hipMemcpy(h_Cor, d_Cor, sizeof(float) * corSize, hipMemcpyDeviceToHost);

	//	int ja = 0;
	//	float *pp = new float[300000];
	//	for(int i = 0, temp = 0; i < corSize; i++)
	//	{
	//		temp = 1000.0*h_Cor[i];
	//		if(abs((int) temp) > 998)
	//		{
	//			pp[ja++] = i;
	////			ja++;
	//		}
	//	}
	//	cout<<"After addition:"<<ja<<endl;
	////create file for Lovain analysis: P1 X Pn....; P2 X Pn-1....
	//for(int i = 0; i < corSize; i++)
	//{
	//	h_final[i].addrss = h_PL[i].win;
	//	h_final[i].Xaddrss = h_PL[i + j].win;
	//	h_final[i].XcorVal = h_Cor[j];
	//}
	//	cout<<ja<<endl;

	//write out the data to a file
	//FILE *fpw;
	//char filew[512];
	//sprintf(filew,"%s.pair.txt","cor_weights");
	//if ((fpw = fopen(filew,"w"))==NULL)
	//	{
	//	printf("cannot open file\n");
	//	}

	//fprintf(fpw, "Pixel No\tNode 1\tNode 2\tScale\n");
	//for(int i = 0; i < corSize; i++)
	//	{
	//	Floc1 = h_Cor[i].loc_Wind1 % frames;
	//	Floc2 = h_Cor[i].loc_Wind2 % frames;
	//	Xloc1 = floor((h_Cor[i].loc_Wind1/imageY));
	//	Yloc1 = (h_Cor[i].loc_Wind1-Floc1) - (Xloc1*imageY);
	//	Xloc1 = Xloc1 + 1;
	//	if (~Yloc1)
	//		Yloc1=imageX;
	//	Xloc2 = floor((h_Cor[i].loc_Wind2-Floc2)/imageY);
	//	Yloc2 = (h_Cor[i].loc_Wind2-Floc2) - (Xloc2*imageY);
	//	if (~Yloc2)
	//		Yloc2=imageX;
	//	fprintf(fpw, "%d\t%d\t%d\t%f\n",Xloc2, Yloc2, Floc2, h_Cor[i].loc_corrCoef);
	//	}
	FILE *fpw;
	char filew[512];
	sprintf(filew,"%s.pair.txt","cor_weights");
	if ((fpw = fopen(filew,"w"))==NULL)
		{
		printf("cannot open file\n");
		}
//	fprintf(fpw, "Pt#1\tFrm#\t\Pt#2\t\Frm#\tXcorr\n");
	fprintf(fpw, "Frm#\tPt#1\t\Pt#2\t\Xcorr\n");
	for(int i = 0; i < corSize; i++)
		{
		Floc1 = h_Cor[i].loc_Wind1 % frames;
		Floc2 = h_Cor[i].loc_Wind2 % frames;
		Yloc1 = (h_Cor[i].loc_Wind1-Floc1)/frames;
		Yloc2 = (h_Cor[i].loc_Wind2-Floc2)/frames;
		//Xloc1 = Yloc1%imageX;
		//Yloc1 = (Yloc1 - Xloc1)/imageX;
		//Xloc2 = Yloc2%imageX;
		//Yloc2 = (Yloc2 - Xloc2)/imageX;
		//Xloc1 += 1;
		//Xloc2 += 1;
		Yloc1 += 1;
		Yloc2 += 1;
		//if (~Yloc1)
		//	Yloc1=imageX;
		//Xloc2 = floor((h_Cor[i].loc_Wind2-Floc2)/imageY);
		//Yloc2 = (h_Cor[i].loc_Wind2-Floc2) - (Xloc2*imageY);

		if(Floc1 == Floc2)
			fprintf(fpw, "%d\t%d\t%d\t%f\n",Floc1,Yloc1, Yloc2,  h_Cor[i].loc_corrCoef);
		//if (~Yloc2)
		//	Yloc2=imageX;
		//		fprintf(fpw, "%d\t%d\t%d\t%d\t%f\n",Yloc1, Floc1,Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		printf("%d\t%d\t%d\t%d\t%f\n",Yloc1, Floc1,Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		fprintf(fpw, "%d\t%d\t%d\t%d\t%d\t%d\t%f\n",Xloc1, Yloc1, Floc1, Xloc2, Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		fprintf(fpw, "Pt1(x,y,f) = %d,%d,%d Pt2(x,y,f) = %d,%d,%d Xcorr = %f\n",Xloc1, Yloc1, Floc1, Xloc2, Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		}

	fclose(fpw);
	//		delete[] h_Cor;
	//		hipFree(d_Cor);
	hipHostFree(h_Cor);
	hipFree(d_PL);
	delete[] h_PL;
	return 0;
	}
